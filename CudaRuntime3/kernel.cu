#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <map>
#include <string>
#include <random>
#include <iostream>
#include <chrono>
__global__ void gpufreq(const char* text, int* res, int count)
{
    int threadid = blockIdx.x * blockDim.x + threadIdx.x;
    if (threadid < count)
    {
        int id = text[threadid];
        atomicAdd(&res[id], 1);
    }
}
void cpufreq(const std::vector<char> h_input, std::vector <int> &hh_ascii)
{
    for (auto v : h_input)
        hh_ascii[(int)v]++;
}
void generateString(std::vector<char> &input, size_t count)
{
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<> rg(65, 122);
    for (int i = 0; i < count; i++)
        input[i] = rg(gen);
}
bool checkOk(std::vector<int> vec1, std::vector<int> vec2)
{
    bool check = 1;
    for (int i = 65; i < 122; i++)
    {
        if (vec1[i] != vec2[i])
            check = 0;
    }
    return check;
}
int main()
{
    int len;
    std::cout << "enter count of letters:\n";
    std::cin >> len;
    std::vector<char> h_input(len);
    generateString(h_input, len);
    //for (auto v : h_input)
    //    std::cout << v;
    std::vector<int> h_ascii(256,0);
    std::vector<int> hh_ascii(256, 0);
    char* d_input;
    int *d_ascii;

    int BLOCK_SIZE = 32;
    dim3 dimGrid(ceil(double(len)/ double(BLOCK_SIZE)));
    dim3 dimBlock(BLOCK_SIZE);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    //MEM ALLOC
    hipMalloc(&d_input, sizeof(char) * len);
    hipMalloc(&d_ascii, sizeof(int) * 256);
    hipEventRecord(start);
    hipMemcpy(d_input, h_input.data(), len*sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_ascii, h_ascii.data(), 256*sizeof(int), hipMemcpyHostToDevice);

    //GPU CALL
    
    gpufreq <<< dimGrid, dimBlock >>> (d_input, d_ascii, len);
    hipDeviceSynchronize();
    

    hipMemcpy(h_ascii.data(), d_ascii, 256 * sizeof(int), hipMemcpyDeviceToHost);


    hipEventRecord(stop);


    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "\ngpu milliseconds elapsed: " << milliseconds;
    std::cout << '\n';

    hipEventDestroy(start);
    hipEventDestroy(stop);

    for (int i = 65; i < 122; i++)
        std::cout <<char(i)<<": " << h_ascii[i]<< "\n";
    int control_sum = 0;
    for (auto v : h_ascii)
        control_sum += v;
    std::cout << "\ncontrol sum: " << control_sum;

    //CPU CALL
    auto begin = std::chrono::steady_clock::now();
    cpufreq(h_input, hh_ascii);
    auto end = std::chrono::steady_clock::now();
    auto elapsed_ms = std::chrono::duration_cast<std::chrono::milliseconds>(end - begin);
    std::cout << "\ncpu milliseconds elapsed: " << elapsed_ms.count();
    control_sum = 0;
    for (auto v : hh_ascii)
        control_sum += v;
    std::cout << '\n';
    for (int i = 65; i < 122; i++)
        std::cout << char(i) << ": " << hh_ascii[i] << "\n";
    std::cout << "\ncontrol sum: " << control_sum;

    if (checkOk(h_ascii, hh_ascii))
        std::cout << "\nall ok";
    else std::cout << "\nnot ok";
}
